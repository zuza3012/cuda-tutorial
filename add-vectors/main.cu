#include "hip/hip_runtime.h"
#include <stdio.h>
#define N 10

__global__ void add(int *a, int *b, int *c){
    int tid = blockIdx.x;
    if(tid < N){
        c[tid] = a[tid] + b[tid];   
    }
}


int main(void){

    int nDevices;

    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n",
            prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
            prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
            2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    }

    
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;
    

    // memory allocation on GPU
    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_b, N * sizeof(int));
    hipMalloc((void**)&dev_c, N * sizeof(int));

    // insert data to arays

    for(int i = 0; i < N; i++){
        a[i] = 1;
        b[i] = 2;
    }
    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
    add<<<N,1>>>(dev_a, dev_b, dev_c);
    hipDeviceSynchronize();
    // copy data to GPU
    hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);
    
   // see result
    for(int i = 0; i < N; i++){
        printf("%d + %d = %d \n", a[i], b[i], c[i]);
    }
    

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
   
    
    return 0;
}