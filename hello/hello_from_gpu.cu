
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel( void ) { 
    printf("Hello from gpu!\n");
}

int main( void ) {

    kernel<<<1,1>>>();
    hipDeviceSynchronize();
    //printf( "Hello, World!\n" );

    return 0; 

}
